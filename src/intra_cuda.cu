#include <hip/hip_runtime.h>

#ifndef __CUDACC__
#define __CUDACC__
#include <device_functions.h>
#endif


#include <iostream>

const int BUFFER_SIZE = 10;

__global__ void cudaReduce(int *const recvbuffs[], int *recvbuff, size_t count,
                           int root, int ndev) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < count) {
    int sum = 0;
    for (int i = 0; i < ndev; i++) {
      sum += recvbuffs[i][index];
    }
    recvbuff[index] = sum;
  }
  __syncthreads();
}

int intraReduce(int *const sendbuff[], int *recvbuff, size_t count, int root,
                int *devs, int ndev) {
  hipStream_t streams[ndev];
  for (int i = 0; i < ndev; i++) {
    hipSetDevice(devs[i]);
    hipStreamCreate(&streams[i]);
  }
  int **recvbuffs;
  hipMallocManaged(&recvbuffs, ndev * sizeof(int *));
  hipSetDevice(devs[root]);
  for (int i = 0; i < ndev; i++) {
    hipMallocManaged(&recvbuffs[i], count * sizeof(int));
  }
  for (int i = 0; i < ndev; i++) {
    if (i != root) {
      hipMemcpyPeerAsync(recvbuffs[i], root, sendbuff[i], devs[i],
                          count * sizeof(int), streams[i]);
    } else {
      hipMemcpyAsync(recvbuffs[i], sendbuff[i], count * sizeof(int),
                      hipMemcpyDeviceToDevice, streams[i]);
    }
  }
  for (int i = 0; i < ndev; i++) {
    hipSetDevice(devs[i]);
    hipStreamSynchronize(streams[i]);
  }
  hipDeviceSynchronize();
  // for (int i = 0; i < ndev; i++) {
  //   for (int j = 0; j < count; j++) {
  //     std::cout << recvbuffs[i][j] << " ";
  //   }
  //   std::cout << std::endl;
  // }
  // std::cout << std::endl;
  hipSetDevice(devs[root]);
  int blockSize = 256;
  int numBlocks = (count + blockSize - 1) / blockSize;
  cudaReduce<<<numBlocks, blockSize>>>(recvbuffs, recvbuff, count, root, ndev);
  hipDeviceSynchronize();
  for (int i = 0; i < ndev; i++) {
    if (i != root) {
      hipFree(recvbuffs[i]);
    }
  }
  for (int i = 0; i < ndev; i++) {
    hipSetDevice(devs[i]);
    hipStreamDestroy(streams[i]);
  }
  return 0;
}

int intraBroadcast(int *sendbuff, int *const recvbuff[], size_t count, int root,
                   int *devs, int ndev) {
  hipStream_t streams[ndev];
  for (int i = 0; i < ndev; i++) {
    hipSetDevice(devs[i]);
    hipStreamCreate(&streams[i]);
  }
  for (int i = 0; i < ndev; i++) {
    hipMemcpyPeerAsync(recvbuff[i], devs[i], sendbuff, root,
                        count * sizeof(int), streams[i]);
  }
  for (int i = 0; i < ndev; i++) {
    hipSetDevice(devs[i]);
    hipStreamSynchronize(streams[i]);
  }
  hipDeviceSynchronize();
  for (int i = 0; i < ndev; i++) {
    hipSetDevice(devs[i]);
    hipStreamDestroy(streams[i]);
  }
  return 0;
}

int main(int argc, char *argv[]) {
  int ndev;
  hipGetDeviceCount(&ndev);

  int *data, *output, *buffer[ndev], devs[ndev];
  for (int i = 0; i < ndev; i++) {
    devs[i] = i;
  }

  for (int i = 0; i < ndev; i++) {
    hipSetDevice(i);
    hipMallocManaged(&buffer[i], BUFFER_SIZE * sizeof(int));
    hipMemset(buffer[i], 0, BUFFER_SIZE * sizeof(int));
  }
  hipSetDevice(0);
  hipMallocManaged(&data, BUFFER_SIZE * sizeof(int));
  hipMemset(data, 0, BUFFER_SIZE * sizeof(int));
  hipMallocManaged(&output, BUFFER_SIZE * sizeof(int));
  hipMemset(output, 0, BUFFER_SIZE * sizeof(int));

  for (int i = 0; i < BUFFER_SIZE; i++) {
    data[i] = i;
  }

  intraBroadcast(data, buffer, BUFFER_SIZE, 0, devs, ndev);

  // for (int i = 0; i < ndev; i++) {
  //   for (int j = 0; j < BUFFER_SIZE; j++) {
  //     std::cout << buffer[i][j] << " ";
  //   }
  //   std::cout << std::endl;
  // }

  intraReduce(buffer, output, BUFFER_SIZE, 0, devs, ndev);

  for (int i = 0; i < BUFFER_SIZE; i++) {
    std::cout << output[i] << " ";
  }
  std::cout << std::endl;

  hipSetDevice(0);
  hipFree(data);
  hipFree(output);
  for (int i = 0; i < ndev; i++) {
    hipSetDevice(i);
    hipFree(buffer[i]);
  }

  return 0;
}
