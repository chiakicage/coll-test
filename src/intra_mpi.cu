#include <mpi.h>
#include <nccl.h>

#include <iostream>

const int BUFFER_SIZE = 10;

int intraReduce(const int* sendbuff, int* recvbuff, size_t count, int root,
                ncclComm_t comm, hipStream_t stream) {
  ncclReduce(sendbuff, recvbuff, count, ncclInt, ncclSum, root, comm, stream);
  hipStreamSynchronize(stream);
  return 0;
}

int intraBroadcast(const int* sendbuff, int* recvbuff, size_t count, int root,
                   ncclComm_t comm, hipStream_t stream) {
  ncclBroadcast(sendbuff, recvbuff, count, ncclInt, root, comm, stream);
  hipStreamSynchronize(stream);
  return 0;
}

int main(int argc, char* argv[]) {
  MPI_Init(&argc, &argv);
  int rank, size;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &size);

  int count;
  hipGetDeviceCount(&count);
  if (size != count) {
    std::cout << "size != nDev" << std::endl;
    return 0;
  }

  ncclUniqueId id;
  ncclComm_t comm;
	int *data, *output, *buffer;
  hipStream_t s;

  if (rank == 0) ncclGetUniqueId(&id);
  MPI_Bcast((void*)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
  std::cout << "rank: " << rank << std::endl;

  hipSetDevice(rank);
  hipStreamCreate(&s);
  hipMallocManaged(&buffer, BUFFER_SIZE * sizeof(int));
  hipMemset(buffer, 0, BUFFER_SIZE * sizeof(int));

  if (rank == 0) {
    hipMallocManaged(&data, BUFFER_SIZE * sizeof(int));
    hipMemset(data, 0, BUFFER_SIZE * sizeof(int));
    hipMallocManaged(&output, BUFFER_SIZE * sizeof(int));
    hipMemset(output, 0, BUFFER_SIZE * sizeof(int));

    for (int i = 0; i < BUFFER_SIZE; i++) {
      data[i] = i;
    }
  }

  ncclCommInitRank(&comm, size, id, rank);

  intraBroadcast(data, buffer, BUFFER_SIZE, 0, comm, s);

  intraReduce(buffer, output, BUFFER_SIZE, 0, comm, s);


  hipStreamSynchronize(s);

  if (rank == 0) {
    for (int i = 0; i < BUFFER_SIZE; i++) {
      std::cout << output[i] << std::endl;
    }
  }

  hipStreamDestroy(s);

  ncclCommDestroy(comm);

  MPI_Finalize();

  return 0;
}
