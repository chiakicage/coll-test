#include <nccl.h>
#include <iostream>
#include <mpi.h>

const int BUFFER_SIZE = 10;
const int nDev = 4;


int intraReduce(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t* comms) {
	hipStream_t streams[nDev];
	for (int i = 0; i < nDev; i++) {
		hipSetDevice(i);
		hipStreamCreate(&streams[i]);
	}

	ncclGroupStart();
	ncclReduce(sendbuff, recvbuff, count, datatype, op, root, comm, stream);
	ncclGroupEnd();
	return 0;
}

int intraBroadcast(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype, int root, ncclComm_t comm, hipStream_t stream) {
	ncclGroupStart();
	ncclBroadcast(sendbuff, recvbuff, count, datatype, root, comm, stream);
	ncclGroupEnd();
	return 0;
}

int main(int argc, char* argv[]) {
	int nDev;
	hipGetDeviceCount(&nDev);

	

	ncclComm_t comms[nDev];
	ncclCommInitAll(comms, nDev, NULL);

	int* sendbuf[nDev];
	int* recvbuf[nDev];

	for (int i = 0; i < nDev; i++) {
		hipSetDevice(i);
		hipMallocManaged(&sendbuf[i], BUFFER_SIZE * sizeof(int));
		// hipMemset(sendbuf[i], 1, BUFFER_SIZE * sizeof(int));
		for (int j = 0; j < BUFFER_SIZE; j++) {
			sendbuf[i][j] = j;
		}
		hipMallocManaged(&recvbuf[i], BUFFER_SIZE * sizeof(int));
		hipMemset(recvbuf[i], 0, BUFFER_SIZE * sizeof(int));
	}

	// ncclCommFinalize(comm);
	// ncclCommDestroy(comm);
	for (int i = 0; i < nDev; i++) {
		hipSetDevice(i);
		hipStreamSynchronize(streams[i]);
	}

	int sum = 0;
	hipSetDevice(1);
	for (int i = 0; i < BUFFER_SIZE; i++) {
		// sum += recvbuf[0][i];
		std::cout << recvbuf[1][i] << std::endl;
	}
	std::cout << "sum: " << sum << std::endl;
	std::cout << "done" << std::endl;

	for (int i = 0; i < nDev; i++) {
		hipSetDevice(i);
		hipFree(sendbuf[i]);
		hipFree(recvbuf[i]);
	}

	for (int i = 0; i < nDev; i++) {
		ncclCommDestroy(comms[i]);
	}

	return 0;
}
