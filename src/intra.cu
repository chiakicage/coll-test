#include <mpi.h>
#include <nccl.h>

#include <iostream>

const int BUFFER_SIZE = 10;
const int nDev = 4;

int intraReduce(int *const sendbuff[], int* recvbuff, size_t count, int root,
                ncclComm_t *comms, hipStream_t *streams) {
  ncclGroupStart();
  for (int i = 0; i < nDev; i++) {
    ncclReduce(sendbuff[i], recvbuff, count, ncclInt, ncclSum, root, comms[i], streams[i]);
  }
  for (int i = 0; i < nDev; i++) {
    hipSetDevice(i);
    hipStreamSynchronize(streams[i]);
  }
  ncclGroupEnd();
  return 0;
}

int intraBroadcast(int* sendbuff, int *const recvbuff[], size_t count, int root,
                   ncclComm_t *comms, hipStream_t *streams) {
  ncclGroupStart();
  for (int i = 0; i < nDev; i++) {
    ncclBroadcast(sendbuff, recvbuff[i], count, ncclInt, root, comms[i], streams[i]);
  }
  for (int i = 0; i < nDev; i++) {
    hipSetDevice(i);
    hipStreamSynchronize(streams[i]);
  }
  ncclGroupEnd();
  return 0;
}

int main(int argc, char* argv[]) {
  


  ncclComm_t comms[nDev];
  int *data, *output, *buffer[nDev];
  hipStream_t s[nDev];

  
  for (int i = 0; i < nDev; i++) {
    hipSetDevice(i);
    hipStreamCreate(&s[i]);
    hipMallocManaged(&buffer[i], BUFFER_SIZE * sizeof(int));
    hipMemset(buffer[i], 0, BUFFER_SIZE * sizeof(int));
  }
  hipSetDevice(0);
  hipMallocManaged(&data, BUFFER_SIZE * sizeof(int));
  hipMemset(data, 0, BUFFER_SIZE * sizeof(int));
  hipMallocManaged(&output, BUFFER_SIZE * sizeof(int));
  hipMemset(output, 0, BUFFER_SIZE * sizeof(int));

  for (int i = 0; i < BUFFER_SIZE; i++) {
    data[i] = i;
  }
 

  ncclCommInitAll(comms, nDev, NULL);


  intraBroadcast(data, buffer, BUFFER_SIZE, 0, comms, s);

  intraReduce(buffer, output, BUFFER_SIZE, 0, comms, s);

  for (int i = 0; i < BUFFER_SIZE; i++) {
    std::cout << output[i] << " ";
  }
  std::cout << std::endl;

  hipSetDevice(0);
  hipFree(data);
  hipFree(output);
  for (int i = 0; i < nDev; i++) {
    hipSetDevice(i);
    hipStreamDestroy(s[i]);
    hipFree(buffer[i]);
    ncclCommDestroy(comms[i]);
  }


  return 0;
}
